// For compiling, use "nvcc -O3"; upon logging on to a CHPC node, "module load cuda" is needed to load "nvcc"


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define threshold 0.0000001
#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4

void checkCUDAError(const char *msg);

const int DSIZE = 1024;
hipEvent_t start, stop;
float tstart, elapsedTime;

// matrix multiply kernel: C = A * B
__global__ void mmul(const double *A, const double *B, double *C, int ds) {
// Enter GPU kernel code body

  int idy = blockDim.x*blockIdx.x+threadIdx.x; // create thread x index
  int idx = blockDim.y*blockIdx.y+threadIdx.y; // create thread y index

  if ((idx < ds) && (idy < ds)){
    double temp = 0;
    for (int k = 0; k < ds; k++)
      temp += A[k*ds+idx] * B[idy*ds+k];   // dot product of row and column
    C[idx*ds+idy] = temp;
  }
}

int main(){

  double *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;

  h_A = new double[DSIZE*DSIZE];
  h_B = new double[DSIZE*DSIZE];
  h_C = new double[DSIZE*DSIZE];
  h_Cref = new double[DSIZE*DSIZE];
  for (i = 0; i < DSIZE*DSIZE; i++){
    h_A[i] = i-1;
    h_B[i] = i+1;
    h_C[i] = 0;
    h_Cref[i] = 0;}

  for (i=0;i<DSIZE;i++)
   for (k=0;k<DSIZE;k++)
    for (j=0;j<DSIZE;j++)
  // h_Cref[i][j] += h_A[k][i]*h_B[j][k];
     h_Cref[i*DSIZE+j] += h_A[k*DSIZE+i]*h_B[j*DSIZE+k];
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(double));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(double));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(double));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(double), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  dim3 block(1,1);  
  dim3 grid(1,1);
  int Bx, By;
  printf("Matrix size: %d\n", DSIZE);
  while(1)
 {
  printf("Specify TB-size-x,TB-size-y: ");
  scanf("%d %d", &Bx,&By);
  if ((Bx==0) or (By==0)) break;
  block.x = Bx;
  block.y = By;
  grid.x = ceil(DSIZE/float(Bx));
  grid.y = ceil(DSIZE/float(By));

  for(int trial=0;trial<5;trial++)
  {
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);
   // Launch kernel
   mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
   checkCUDAError("kernel launch");
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
   hipDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(double), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int i = 0; i < DSIZE*DSIZE; i++) if (fabs((h_C[i] - h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
   printf("<BX=%d,BY=%d>: Trial %d: GFLOPS: %.2f\n",Bx,By,trial,2.0e-6*DSIZE*DSIZE*DSIZE/elapsedTime);
  }
 }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

