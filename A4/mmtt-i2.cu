// For compiling, use "nvcc -O3"; upon logging on to a CHPC node, "module load cuda" is needed to load "nvcc"


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define threshold 0.0000001
#define FIXME1 1
#define FIXME2 2
#define FIXME3 3
#define FIXME4 4

void checkCUDAError(const char *msg);

const int DSIZE = 2048;
hipEvent_t start, stop;
float tstart, elapsedTime;

// matrix multiply kernel: C = A * B
__global__ void mmul(const double *A, const double *B, double *C, int ds) {
// Enter GPU kernel code body

  int i = blockDim.x*blockIdx.x*2+threadIdx.x; // create thread x index
  int j = blockDim.y*blockIdx.y+threadIdx.y; // create thread y index
  //printf("index %d %d\n",i,j);
  if ((i < ds) && (j < ds)){
    double temp = 0;
    double temp1 = 0;
    for (int k = 0; k < ds; k++){
      temp += A[k*ds+j] * B[i*ds+k];   // dot product of row and column
      temp1 += A[k*ds+j] * B[(i+blockDim.x)*ds+k];   // dot product of row and column
    }
      
    C[i+j*ds] = temp;
    C[i+blockDim.x+j*ds] = temp1;
    //printf("index %d %d %d %d, value %f %f\n",i,j,i+j*ds,i+(j+1)*ds,temp,temp1);
  }
}
/*
Matrix size: 2048
Specify TB-size-x,TB-size-y: 8 8
grid dimesion 128 256
<BX=8,BY=8>: Trial 0: GFLOPS: 43.97
<BX=8,BY=8>: Trial 1: GFLOPS: 43.98
<BX=8,BY=8>: Trial 2: GFLOPS: 43.97
<BX=8,BY=8>: Trial 3: GFLOPS: 43.97
<BX=8,BY=8>: Trial 4: GFLOPS: 43.97
Specify TB-size-x,TB-size-y: 16 16
grid dimesion 64 128
<BX=16,BY=16>: Trial 0: GFLOPS: 22.67
<BX=16,BY=16>: Trial 1: GFLOPS: 22.67
<BX=16,BY=16>: Trial 2: GFLOPS: 22.67
<BX=16,BY=16>: Trial 3: GFLOPS: 22.67
<BX=16,BY=16>: Trial 4: GFLOPS: 22.67
Specify TB-size-x,TB-size-y: 32 32
grid dimesion 32 64
<BX=32,BY=32>: Trial 0: GFLOPS: 22.54
<BX=32,BY=32>: Trial 1: GFLOPS: 22.54
<BX=32,BY=32>: Trial 2: GFLOPS: 22.53
<BX=32,BY=32>: Trial 3: GFLOPS: 22.54
<BX=32,BY=32>: Trial 4: GFLOPS: 22.53
*/
int main(){

  double *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;

  h_A = new double[DSIZE*DSIZE];
  h_B = new double[DSIZE*DSIZE];
  h_C = new double[DSIZE*DSIZE];
  h_Cref = new double[DSIZE*DSIZE];
  for (i = 0; i < DSIZE*DSIZE; i++){
    h_A[i] = i-1;
    h_B[i] = i+1;
    h_C[i] = 0;
    h_Cref[i] = 0;}

  for (i=0;i<DSIZE;i++)
   for (k=0;k<DSIZE;k++)
    for (j=0;j<DSIZE;j++)
  // h_Cref[i][j] += h_A[k][i]*h_B[j][k];
     h_Cref[i*DSIZE+j] += h_A[k*DSIZE+i]*h_B[j*DSIZE+k];
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(double));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(double));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(double));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(double), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D failure");

  dim3 block(1,1);  
  dim3 grid(1,1);
  int Bx, By;
  printf("Matrix size: %d\n", DSIZE);
  while(1)
 {
  printf("Specify TB-size-x,TB-size-y: ");
  scanf("%d %d", &Bx,&By);
  if ((Bx==0) or (By==0)) break;
  block.x = Bx;
  block.y = By;
  grid.x = ceil(DSIZE/2/float(Bx));
  grid.y = ceil(DSIZE/float(By));
  printf("grid dimesion %d %d\n",grid.x,grid.y);
  for(int trial=0;trial<5;trial++)
  {
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);
   // Launch kernel
   mmul<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
   checkCUDAError("kernel launch");
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
   hipDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(double), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int i = 0; i < DSIZE*DSIZE; i++) if (fabs((h_C[i] - h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
   printf("<BX=%d,BY=%d>: Trial %d: GFLOPS: %.2f\n",Bx,By,trial,2.0e-6*DSIZE*DSIZE*DSIZE/elapsedTime);
  }
 }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

