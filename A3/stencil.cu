// For compiling, use "nvcc -O3"; upon logging on to a CHPC node, "module load cuda" is needed to load "nvcc"


#include <hip/hip_runtime.h>
#include <stdio.h>

void checkCUDAError(const char *msg);

#include <stdio.h>

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4097*257)
#define FIXME1 32
#define FIXME2 32

// The FIXMEs indicate where code must replace the FIXMEs.
// cx
// The number of input elements is N+2*RADIUS, IN[0:N+2*RADIUS-1]
// Each element of out holds the sum of a set of 2*RADIUS+1 contiguous elements from in
// The sum of contents in in[0:2*RADIUS] is placed in out[0], 
// sum of elements in in[1:2*RADIUS+1] is placed in out[1], etc.

__global__ void stencil_1d(int *in, int *out, int N) 
{
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  //out[idx]=0;
  
  if(idx<N){
      for(int r=-RADIUS;r<=RADIUS;r++)
        out[idx] += in[RADIUS+idx+r];
      //printf("%d\n",out[idx]);
  }
}

int main()
{
  int i,r;
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS], h_ref[NUM_ELEMENTS];
  int *d_in, *d_out;

  // Initialize host data
  for(i = 0; i < (NUM_ELEMENTS + 2*RADIUS); i++ )
    h_in[i] = i; 
  for(i = 0; i < NUM_ELEMENTS; i++)
    h_ref[i] = 0;

  for(i = 0; i < NUM_ELEMENTS; i++)
   for(r = -RADIUS; r <= RADIUS; r++)
    h_ref[i] += h_in[RADIUS+i+r];

  // Allocate space on the device
  hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int));
  hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int));
  checkCUDAError("hipMalloc");

  // Copy input data to device
  hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy");

  // Fix the FIXME's
  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(ceil(NUM_ELEMENTS/float(BLOCK_SIZE)));

  stencil_1d<<< dimGrid,dimBlock >>> (d_in, d_out,NUM_ELEMENTS);
  checkCUDAError("Kernel Launch Error:");

  hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);
  checkCUDAError("hipMalloc");

  for( i = 0; i < NUM_ELEMENTS; ++i )
    if (h_ref[i] != h_out[i])
    {
      printf("ERROR: Mismatch at index %d: expected %d but found %d\n",i,h_ref[i], h_out[i]);
      break;
    }

    if (i== NUM_ELEMENTS) printf("SUCCESS!\n");

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

